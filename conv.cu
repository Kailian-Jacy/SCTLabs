#include <hip/hip_runtime.h>

#include <iomanip>
#include <iostream>
#include <random>

const int kSize = 5000;
const int kKernelSize = 13; // odd

#define InitRandom()                         \
  std::random_device r;                      \
  std::default_random_engine generator(r()); \
  std::uniform_real_distribution<float> distribution(0, 1e3);

void Generate(float *const a, float *const w)
{
#pragma omp parallel for
  for (int i = 0; i < kSize; ++i)
  {
    InitRandom();
    const int j_upperbound = (i + 1) * kSize;
    for (int j = i * kSize; j < j_upperbound; ++j)
      a[j] = distribution(generator);
  }
  {
    InitRandom();
    for (int i = 0; i < kKernelSize * kKernelSize; ++i)
      w[i] = distribution(generator);
  }
}

void Conv(const float *const a, const float *const w, float *const b)
{
#pragma omp parallel for
  for (int i = 0; i < kSize; ++i)
  {
    for (int j = 0; j < kSize; ++j)
    {
      float conv = 0;
      int x = i - kKernelSize / 2, y = j - kKernelSize / 2;
      for (int k = 0; k < kKernelSize; ++k)
      {
        for (int l = 0; l < kKernelSize; ++l)
        {
          if (!(x < 0 || x >= kSize || y < 0 || y >= kSize))
            conv += a[x * kSize + y] * w[k * kKernelSize + l];
          y++;
        }
        x++;
        y -= kKernelSize;
      }
      b[i * kSize + j] = conv;
    }
  }
}

void Check(const float *const a, const float *const w, float *const b)
{
  auto b_std = new float[kSize * kSize];
  Conv(a, w, b_std);
  for (int i = 0; i < kSize * kSize; ++i)
  {
    if (abs(b[i] / b_std[i] - 1) > 1e-3 || isnanf(b[i]) || isinff(b[i]))
    {
      std::cout << "\x1b[31m"
                   "Wrong Answer"
                   "\x1b[0m"
                   " at "
                << i << std::endl;
      std::cout << "expected " << b_std[i] << " but found " << b[i]
                << std::endl;
      delete[] b_std;
      return;
    }
  }
  std::cout << "\x1b[32m"
               "Correct"
               "\x1b[0m"
            << std::endl;

  delete[] b_std;
}

void Output(const float *const a, const float *const w, const float *const b)
{
  for (int i = 0; i < kSize; ++i)
  {
    for (int j = 0; j < kSize; ++j)
      std::cout << std::setw(2) << a[i * kSize + j] << ' ';
    std::cout << std::endl;
  }

  for (int i = 0; i < kKernelSize; ++i)
  {
    for (int j = 0; j < kKernelSize; ++j)
      std::cout << std::setw(2) << w[i * kKernelSize + j] << ' ';
    std::cout << std::endl;
  }

  for (int i = 0; i < kSize; ++i)
  {
    for (int j = 0; j < kSize; ++j)
      std::cout << std::setw(2) << b[i * kSize + j] << ' ';
    std::cout << std::endl;
  }
}

__global__ void ConvGPU(float *ad, float *wd, float *bd, float *tmp, int roundX, int roundY)
{

  int RoundStartingIndex = 25 * roundY + 25 * roundX * kSize;

  float conv = 0;
  int x = blockIdx.x - kKernelSize / 2 + threadIdx.x;
  int y = blockIdx.y - kKernelSize / 2 + threadIdx.y;

  if (!((x + 25 * roundX < 0) || (x + 25 * roundX >= kSize) || (y + 25 * roundY < 0) || (y + 25 * roundY >= kSize)))
  {
    conv = ad[RoundStartingIndex + x * kSize + y] * wd[threadIdx.x * kKernelSize + threadIdx.y];
  }

  unsigned int tidInBlk = threadIdx.x * kKernelSize + threadIdx.y;
  if (tidInBlk >= kKernelSize * kKernelSize)
  {
    return;
  }

  float *idata = tmp + (blockIdx.x * 25 + blockIdx.y) * kKernelSize * kKernelSize;
  idata[tidInBlk] = conv;
  __syncthreads();
  if (tidInBlk == 0)
  {
    for (int i = 1; i < kKernelSize * kKernelSize; i++)
    {
      idata[0] += idata[i];
    }
  }
  __syncthreads();
  bd[RoundStartingIndex + blockIdx.x * kSize + blockIdx.y] = idata[0];
}

int main()
{
  auto a = new float[kSize * kSize];
  auto w = new float[kKernelSize * kKernelSize];
  auto b = new float[kSize * kSize];
  Generate(a, w);

  hipEvent_t start_e, stop_e;
  hipEventCreate(&start_e);
  hipEventCreate(&stop_e);

  hipEventRecord(start_e);

  int tmpKsize = 25;

  // initialize data in device memory.
  float *ad = NULL, *tmp = NULL,
        *wd = NULL, *bd = NULL;
  hipMalloc(&ad, kSize * kSize * sizeof(float));
  hipMalloc(&wd, kKernelSize * kKernelSize * sizeof(float));
  hipMalloc(&bd, kSize * kSize * sizeof(float));
  hipMalloc(&tmp, tmpKsize * tmpKsize * kKernelSize * kKernelSize * sizeof(float));

  hipDeviceSynchronize();

  hipMemcpy(ad, a, kSize * kSize * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(wd, w, kKernelSize * kKernelSize * sizeof(float), hipMemcpyHostToDevice);

  hipDeviceSynchronize();

  // the promoted GPU version.
  dim3 grid(tmpKsize, tmpKsize, 1);
  dim3 block(kKernelSize, kKernelSize, 1);

  for (int i = 0; i < kSize / tmpKsize; i++)
  {
    for (int j = 0; j < kSize / tmpKsize; j++)
    {
      ConvGPU<<<grid, block>>>(ad, wd, bd, tmp, i, j); // for efficiency concern, tmpKsize is hard-coded.
    }
  }
  hipDeviceSynchronize();

  hipMemcpy(b, bd, kSize * kSize * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  Check(a, w, b);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start_e, stop_e);
  std::cout << milliseconds << " milliseconds" << std::endl;
  hipEventDestroy(start_e);
  hipEventDestroy(stop_e);

  // Output(a, w, b);

  delete[] a;
  delete[] w;
  delete[] b;
  return 0;
}
